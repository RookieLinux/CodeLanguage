
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("Hello World from the the GPU, grid%d th%d\n", blockIdx.x, threadIdx.x);
}


int main(void)
{
    hello_from_gpu<<<4, 4>>>();
    hipDeviceSynchronize();

    return 0;
}
